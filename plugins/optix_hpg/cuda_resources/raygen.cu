#include "hip/hip_runtime.h"
#include "camera.h"
#include "raygen.h"

#include "optix/random.h"
#include "optix/random_owl.h"
#include "optix/utils_device.h"

namespace megamol {
namespace optix_hpg {
namespace device {
// OptiX SDK
// Path tracer example

//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

//
// Modified 2021 MegaMol Dev Team
//

// code partially from: https://github.com/UniStuttgart-VISUS/rtxpkd_ldav2020
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// ======================================================================== //
// Modified 2019-2020 VISUS - University of Stuttgart                       //
// ======================================================================== //


//#define RANDVEC3F glm::vec3(rnd(42), rnd(42), rnd(42))
#define RANDVEC3F glm::vec3(rnd(), rnd(), rnd())

inline __device__ glm::vec3 random_in_unit_sphere(owl::common::LCG<16>& rnd) {
    glm::vec3 p;
    do {
        p = 2.0f * RANDVEC3F - glm::vec3(1, 1, 1);
    } while (glm::dot(p, p) >= 1.0f);
    return p;
}

inline __device__ glm::vec4 traceRay(const RayGenData& self, Ray& ray, owl::common::LCG<16>& rnd /*, Random& rnd*/,
    PerRayData& prd, glm::vec4& bg, int maxBounces) {

    unsigned int p0 = 0;
    unsigned int p1 = 0;
    packPointer(&prd, p0, p1);

    glm::vec3 col(1.f);

    for (int depth = 0; true; ++depth) {
        prd.particleID = -1;

        optixTrace(self.world, (const float3&) ray.origin, (const float3&) ray.direction, ray.tmin, ray.tmax, 0,
            (OptixVisibilityMask) -1,
            /*rayFlags     */ OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            /*SBToffset    */ 0,
            /*SBTstride    */ 1,
            /*missSBTIndex */ 0, p0, p1);
        if (prd.particleID == -1) {
            return glm::vec4(col * glm::vec3(0.8f), 1.0f);
        }

        glm::vec3 N = (ray.origin + prd.t * ray.direction) - prd.pos;
        if (glm::dot(N, ray.direction) > 0.f)
            N = -N;
        N = glm::normalize(N);

        if (maxBounces == 0) {
            return glm::vec4(prd.albedo * (.2f + .6f * fabsf(glm::dot(N, ray.direction))), 1.0f);
        }

        col *= prd.albedo;

        if (depth >= maxBounces)
            return glm::vec4(0.1f, 0.1f, 0.1f, 1.0f);

        auto scattered_origin = ray.origin + prd.t * ray.direction;
        auto scattered_direction = N + random_in_unit_sphere(rnd);
        ray = Ray(/* origin   : */ scattered_origin,
            /* direction: */ glm::normalize(scattered_direction),
            /* tmin     : */ 1e-3f,
            /* tmax     : */ 1e+8f);
    }
}

MM_OPTIX_RAYGEN_KERNEL(raygen_program)() {
    // printf("RAYGEN1\n");
    const RayGenData& self = getProgramData<RayGenData>();
    auto const index = optixGetLaunchIndex();
    auto const dim = optixGetLaunchDimensions();
    glm::ivec2 pixelID = glm::ivec2(index.x, index.y);

    if (pixelID.x >= self.fbSize.x)
        return;
    if (pixelID.y >= self.fbSize.y)
        return;
    const int pixelIdx = pixelID.x + self.fbSize.x * pixelID.y;
    const int pixel_index = pixelID.y * dim.x + pixelID.x;

    const FrameState* fs = &self.frameStateBuffer[0];

    /*auto frame_idx = self.colorBufferPtr[pixelIdx].w;
    if (fs->changed) {
        frame_idx = 0.0f;
        self.colorBufferPtr[pixelIdx].w = 0.0f;
    }*/
    // auto const old_col = self.colorBufferPtr[pixelIdx];

    /*float4 old_col;
    surf2Dread(&old_col, self.col_surf, pixelID.x * sizeof(float4), pixelID.y, hipBoundaryModeZero);*/

    //unsigned int seed = tea<16>(pixelID.y * self.fbSize.x + pixelID.x, fs->frameIdx);

    owl::common::LCG<16> rnd_owl(pixel_index, fs->frameIdx);


    glm::vec4 col(0.f);
    glm::vec4 bg = fs->background;

    // printf("RAYGEN FS %f\n", fs->near);

    //auto i = fs->samplesPerPixel;

    float depth = FLT_MAX;

    PerRayData prd;
    for (int s = 0; s < fs->samplesPerPixel; ++s) {
        prd.countDepth = true;
        prd.ray_depth = FLT_MAX;
        float u = -fs->rw + (fs->rw + fs->rw) * (float(pixelID.x) + rnd_owl()) / self.fbSize.x;
        float v = -(fs->th + (-fs->th - fs->th) * (float(pixelID.y) + rnd_owl()) / self.fbSize.y);
        /*float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x) / self.fbSize.x;
        float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y) / self.fbSize.y);*/
        auto ray = generateRay(*fs, u, v);
        col += traceRay(self, ray, rnd_owl, prd, bg, fs->maxBounces);
        depth = fminf(depth, prd.ray_depth);
    }

    col /= (float) fs->samplesPerPixel;
    // col.w = frame_idx + 1;
    //++col.w;

    //if (fs->frameIdx > 0) {
    //    const float a = 1.0f / static_cast<float>(fs->frameIdx + 1);
    //    col = lerp(glm::vec4(static_cast<float>(old_col.x), static_cast<float>(old_col.y),
    //                   static_cast<float>(old_col.z), static_cast<float>(old_col.w)),
    //        col, a);
    //    // col.w = frame_idx + 1;
    //}

    if (depth < FLT_MAX) {
        depth = (fs->depth_params.z / depth) - (fs->depth_params.x);
        depth = 0.5f * (depth + 1.0f);
    } else {
        depth = 1.f;
        col = bg;
    }
    //surf2Dwrite(depth, self.depth_surf, pixelID.x * sizeof(float), pixelID.y, hipBoundaryModeZero);


    if (fs->frameIdx > 0 && fs->accumulate)
        col += self.accumBuffer[pixelIdx];
    self.accumBuffer[pixelIdx] = col;

    if (fs->accumulate)
        col /= float(fs->frameIdx + 1);
    surf2Dwrite(make_float4(col.r, col.g, col.b, col.a), self.col_surf, pixelID.x * sizeof(float4), pixelID.y,
        hipBoundaryModeZero);
    /*surf2Dwrite(make_float4(1, 1, 1, 1), self.col_surf, pixelID.x * sizeof(float4), pixelID.y,
        hipBoundaryModeZero);*/


    //surf2Dwrite(1, self.depth_surf, pixelID.x * sizeof(float), pixelID.y, hipBoundaryModeZero);
}
} // namespace device
} // namespace optix_hpg
} // namespace megamol
