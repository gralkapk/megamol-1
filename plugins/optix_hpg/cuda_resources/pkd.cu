#include "hip/hip_runtime.h"
// PKD implementation:
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// ======================================================================== //
// Modified 2019-2020 VISUS - University of Stuttgart                       //
// ======================================================================== //

#include "pkd.h"

#include "optix/utils_device.h"

namespace megamol {
namespace optix_hpg {
namespace device {

inline __device__ bool clipToBounds(const Ray& ray, const box3f& bounds, float& t0, float& t1) {
    glm::vec3 t_lower = (bounds.lower - ray.origin) / ray.direction;
    glm::vec3 t_upper = (bounds.upper - ray.origin) / ray.direction;

    glm::vec3 t_min3 = min(t_lower, t_upper);
    glm::vec3 t_max3 = max(t_lower, t_upper);

    t0 = fmaxf(ray.tmin, glm::max(t_min3.x, glm::max(t_min3.y, t_min3.z)));
    t1 = fminf(ray.tmax, glm::min(t_max3.x, glm::min(t_max3.y, t_max3.z)));
    return t0 < t1;
}

inline __device__ bool intersectSphere(const PKDParticle& particle, const float particleRadius, const Ray& ray, float& hit_t) {
    // Raytracing Gems Intersection Code (Chapter 7)
    //const glm::vec3 pos = glm::vec3(particle.x, particle.y, particle.z);
    const glm::vec3 oc = ray.origin - particle.pos;
    const float sqrRad = particleRadius * particleRadius;

    // const float  a = dot(ray.direction, ray.direction);
    const float b = glm::dot(-oc, ray.direction);
    const glm::vec3 temp = oc + b * ray.direction;
    const float delta = sqrRad - glm::dot(temp, temp);

    if (delta < 0.0f)
        return false;

    const float c = glm::dot(oc, oc) - sqrRad;
    const float q = b + copysignf(sqrtf(delta), b);

    {
        float temp = fminf(c / q, q);
        if (temp < hit_t && temp > ray.tmin) {
            hit_t = temp;
            return true;
        }
    }

    return false;
}

struct StackEntry {
    float t0, t1;
    unsigned int nodeID;
};

MM_OPTIX_INTERSECTION_KERNEL(pkd_intersect)() {
    const auto& self = getProgramData<PKDGeoData>();

    float t0, t1;
    {
        auto const ray =
            Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
        if (!clipToBounds(ray, self.worldBounds, t0, t1))
            return;
    }
    auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0), fminf(optixGetRayTmax(), t1));

    int nodeID = 0;
    float tmp_hit_t = t1;
    int tmp_hit_primID = -1;

    enum { STACK_DEPTH = 32 };
    StackEntry stackBase[STACK_DEPTH];
    StackEntry* stackPtr = stackBase;

    const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
    const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
    const float rdir[3] = {
        (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
        (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
        (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
    };
    unsigned int const numParticles = self.particleCount;
    float const particleRadius = self.radius;

    while (1) {
        // while we have anything to traverse ...

        while (1) {
            // while we can go down
            const PKDParticle& particle = self.particleBufferPtr[nodeID];
            int const dim = particle.dim;

            const float t_slab_lo = (particle.pos[dim] - particleRadius - org[dim]) * rdir[dim];
            const float t_slab_hi = (particle.pos[dim] + particleRadius - org[dim]) * rdir[dim];

            const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
            const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

            // -------------------------------------------------------
            // compute potential sphere interval, and intersect if necessary
            // -------------------------------------------------------
            const float sphere_t0 = fmaxf(t0, t_slab_nr);
            const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

            if (sphere_t0 < sphere_t1) {
                if (intersectSphere(particle, particleRadius, ray, tmp_hit_t))
                    tmp_hit_primID = nodeID;
            }

            // -------------------------------------------------------
            // compute near and far side intervals
            // -------------------------------------------------------
            const float nearSide_t0 = t0;
            const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

            const float farSide_t0 = fmaxf(t0, t_slab_nr);
            const float farSide_t1 = fminf(t1, tmp_hit_t);


            // -------------------------------------------------------
            // logic
            // -------------------------------------------------------
            const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
            const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

            const bool nearSide_valid = nearSide_nodeID < numParticles;
            const bool farSide_valid = farSide_nodeID < numParticles;

            const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
            const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

            if (!(need_nearSide || need_farSide))
                break; // pop ...

            if (need_nearSide && need_farSide) {
                stackPtr->t0 = farSide_t0;
                stackPtr->t1 = farSide_t1;
                stackPtr->nodeID = farSide_nodeID;
                ++stackPtr;

                nodeID = nearSide_nodeID;
                t0 = nearSide_t0;
                t1 = nearSide_t1;
                continue;
            }

            nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
            t0 = need_nearSide ? nearSide_t0 : farSide_t0;
            t1 = need_nearSide ? nearSide_t1 : farSide_t1;
        }
        // -------------------------------------------------------
        // pop
        // -------------------------------------------------------
        while (1) {
            if (stackPtr == stackBase) {
                // can't pop any more - done.
                if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                    optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                }
                return;
            }
            --stackPtr;
            t0 = stackPtr->t0;
            t1 = fminf(stackPtr->t1, tmp_hit_t);
            nodeID = stackPtr->nodeID;
            if (t1 <= t0)
                continue;
            break;
        }
    }
}


MM_OPTIX_CLOSESTHIT_KERNEL(pkd_closesthit)() {
    //const int primID = optixGetPrimitiveIndex();
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<PKDGeoData>();

    prd.particleID = primID;
    const PKDParticle& particle = self.particleBufferPtr[primID];
    prd.pos = particle.pos;
    glm::vec3 geo_col = glm::vec3(self.globalColor);
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]);
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(pkd_closesthit_occlusion)() {
    optixSetPayload_0(1);
}


MM_OPTIX_BOUNDS_KERNEL(pkd_bounds)
(const void* geomData, const float* radData, float radius, box3f& primBounds, const unsigned int primID) {}


MM_OPTIX_INTERSECTION_KERNEL(treelets_intersect)
() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = getProgramData<TreeletsGeoData>();
    const auto treelet = self.treeletBufferPtr[treeletID];

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        {
            auto const ray =
                Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
            if (!clipToBounds(ray, treelet.bounds, t0, t1))
                return;
        }
        auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0), fminf(optixGetRayTmax(), t1));
        
        int nodeID = 0;
        float tmp_hit_t = ray.tmax;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };
        StackEntry stackBase[STACK_DEPTH];
        StackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;
                const PKDParticle& particle = self.particleBufferPtr[particleID];
                int const dim = particle.dim;

                const float t_slab_lo = (particle.pos[dim] - self.radius - org[dim]) * rdir[dim];
                const float t_slab_hi = (particle.pos[dim] + self.radius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(particle, self.radius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;
                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);
                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit)
() {
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<TreeletsGeoData>();

    prd.particleID = primID;
    const PKDParticle& particle = self.particleBufferPtr[primID];
    prd.pos = particle.pos;
    glm::vec3 geo_col = glm::vec3(self.globalColor);
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]);
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit_occlusion)
() {
    optixSetPayload_0(1);
}


MM_OPTIX_BOUNDS_KERNEL(treelets_bounds)
(const void* geomData, const float* radData, float radius, box3f& primBounds, const unsigned int primID) {}

} // namespace device
} // namespace optix_hpg
} // namespace megamol
