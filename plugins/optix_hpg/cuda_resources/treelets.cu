#include "hip/hip_runtime.h"
#include "pkd.h"

#include "optix/utils_device.h"
#include "pkd_utils.h"
#include "ptx_util.h"

#include "datatools/PKDUtils.h"

namespace megamol {
namespace optix_hpg {
namespace device {

MM_OPTIX_INTERSECTION_KERNEL(treelets_intersect_flat)() {
    const int treeletID = optixGetPrimitiveIndex();

    const auto& self = getProgramData<TreeletsGeoData>();

    const auto treelet = self.treeletBufferPtr[treeletID];

    auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
    float t0, t1;
    if (!clipToBounds(ray, treelet.bounds, t0, t1))
        return;

    float tmp_hit_t = ray.tmax;
    int tmp_hit_primID = -1;

    for (unsigned int i = treelet.begin; i < treelet.end; ++i) {
        const glm::vec3& particle = self.particleBufferPtr[i];
        if (intersectSphere(particle, self.radius, ray, tmp_hit_t)) {
            tmp_hit_primID = i;
        }
    }

    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
    }
}


MM_OPTIX_INTERSECTION_KERNEL(treelets_intersect)
() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = getProgramData<TreeletsGeoData>();
    const auto treelet = self.treeletBufferPtr[treeletID];

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        //{
        auto const ray =
            Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
        if (!clipToBounds(ray, treelet.bounds, t0, t1))
            return;
        //}
        /*auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0),
            fminf(optixGetRayTmax(), t1));*/

        int nodeID = 0;
        float tmp_hit_t = ray.tmax;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };
        StackEntry stackBase[STACK_DEPTH];
        StackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;
                const glm::vec3& particle = self.particleBufferPtr[particleID];
                int const dim = getDim(particle);

                const float t_slab_lo = (particle[dim] - self.radius - org[dim]) * rdir[dim];
                const float t_slab_hi = (particle[dim] + self.radius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(particle, self.radius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;
                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);
                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit)
() {
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<TreeletsGeoData>();

    prd.particleID = primID;
    const glm::vec3& particle = self.particleBufferPtr[primID];
    prd.pos = particle;
    glm::vec3 geo_col = glm::vec3(self.globalColor) / 255.f;
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]) / 255.f;
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit_occlusion)
() {
    optixSetPayload_0(1);
}


MM_OPTIX_BOUNDS_KERNEL(treelets_bounds)
(const void* geomData, const float* radData, float radius, box3f& primBounds, const unsigned int primID) {}

} // namespace device
} // namespace optix_hpg
} // namespace megamol
