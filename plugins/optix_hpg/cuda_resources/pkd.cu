#include "hip/hip_runtime.h"
// PKD implementation:
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// ======================================================================== //
// Modified 2019-2020 VISUS - University of Stuttgart                       //
// ======================================================================== //

#include "pkd.h"

#include "optix/utils_device.h"
#include "pkd_utils.h"

namespace megamol {
namespace optix_hpg {
namespace device {

inline __device__ bool clipToBounds(const Ray& ray, const box3f& bounds, float& t0, float& t1) {
    glm::vec3 t_lower = (bounds.lower - ray.origin) / ray.direction;
    glm::vec3 t_upper = (bounds.upper - ray.origin) / ray.direction;

    glm::vec3 t_min3 = min(t_lower, t_upper);
    glm::vec3 t_max3 = max(t_lower, t_upper);

    t0 = fmaxf(ray.tmin, glm::max(t_min3.x, glm::max(t_min3.y, t_min3.z)));
    t1 = fminf(ray.tmax, glm::min(t_max3.x, glm::min(t_max3.y, t_max3.z)));
    return t0 < t1;
}

inline __device__ bool intersectSphere(
    const PKDParticle& particle, const float particleRadius, const Ray& ray, float& hit_t) {
    // Raytracing Gems Intersection Code (Chapter 7)
    //const glm::vec3 pos = glm::vec3(particle.x, particle.y, particle.z);
    const glm::vec3 oc = ray.origin - particle.pos;
    const float sqrRad = particleRadius * particleRadius;

    // const float  a = dot(ray.direction, ray.direction);
    const float b = glm::dot(-oc, ray.direction);
    const glm::vec3 temp = oc + b * ray.direction;
    const float delta = sqrRad - glm::dot(temp, temp);

    if (delta < 0.0f)
        return false;

    const float c = glm::dot(oc, oc) - sqrRad;
    const float q = b + copysignf(sqrtf(delta), b);

    {
        float temp = fminf(c / q, q);
        if (temp < hit_t && temp > ray.tmin) {
            hit_t = temp;
            return true;
        }
    }

    return false;
}

struct StackEntry {
    float t0, t1;
    unsigned int nodeID;
};

MM_OPTIX_INTERSECTION_KERNEL(pkd_intersect)() {
    const auto& self = getProgramData<PKDGeoData>();

    float t0, t1;
    {
        auto const ray =
            Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
        if (!clipToBounds(ray, self.worldBounds, t0, t1))
            return;
    }
    auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0),
        fminf(optixGetRayTmax(), t1));

    int nodeID = 0;
    float tmp_hit_t = t1;
    int tmp_hit_primID = -1;

    enum { STACK_DEPTH = 32 };
    StackEntry stackBase[STACK_DEPTH];
    StackEntry* stackPtr = stackBase;

    const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
    const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
    const float rdir[3] = {
        (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
        (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
        (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
    };
    unsigned int const numParticles = self.particleCount;
    float const particleRadius = self.radius;

    while (1) {
        // while we have anything to traverse ...

        while (1) {
            // while we can go down
            const PKDParticle& particle = self.particleBufferPtr[nodeID];
            int const dim = particle.dim;

            const float t_slab_lo = (particle.pos[dim] - particleRadius - org[dim]) * rdir[dim];
            const float t_slab_hi = (particle.pos[dim] + particleRadius - org[dim]) * rdir[dim];

            const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
            const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

            // -------------------------------------------------------
            // compute potential sphere interval, and intersect if necessary
            // -------------------------------------------------------
            const float sphere_t0 = fmaxf(t0, t_slab_nr);
            const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

            if (sphere_t0 < sphere_t1) {
                if (intersectSphere(particle, particleRadius, ray, tmp_hit_t))
                    tmp_hit_primID = nodeID;
            }

            // -------------------------------------------------------
            // compute near and far side intervals
            // -------------------------------------------------------
            const float nearSide_t0 = t0;
            const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

            const float farSide_t0 = fmaxf(t0, t_slab_nr);
            const float farSide_t1 = fminf(t1, tmp_hit_t);


            // -------------------------------------------------------
            // logic
            // -------------------------------------------------------
            const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
            const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

            const bool nearSide_valid = nearSide_nodeID < numParticles;
            const bool farSide_valid = farSide_nodeID < numParticles;

            const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
            const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

            if (!(need_nearSide || need_farSide))
                break; // pop ...

            if (need_nearSide && need_farSide) {
                stackPtr->t0 = farSide_t0;
                stackPtr->t1 = farSide_t1;
                stackPtr->nodeID = farSide_nodeID;
                ++stackPtr;

                nodeID = nearSide_nodeID;
                t0 = nearSide_t0;
                t1 = nearSide_t1;
                continue;
            }

            nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
            t0 = need_nearSide ? nearSide_t0 : farSide_t0;
            t1 = need_nearSide ? nearSide_t1 : farSide_t1;
        }
        // -------------------------------------------------------
        // pop
        // -------------------------------------------------------
        while (1) {
            if (stackPtr == stackBase) {
                // can't pop any more - done.
                if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                    optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                }
                return;
            }
            --stackPtr;
            t0 = stackPtr->t0;
            t1 = fminf(stackPtr->t1, tmp_hit_t);
            nodeID = stackPtr->nodeID;
            if (t1 <= t0)
                continue;
            break;
        }
    }
}


MM_OPTIX_CLOSESTHIT_KERNEL(pkd_closesthit)() {
    //const int primID = optixGetPrimitiveIndex();
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<PKDGeoData>();

    prd.particleID = primID;
    const PKDParticle& particle = self.particleBufferPtr[primID];
    prd.pos = particle.pos;
    glm::vec3 geo_col = glm::vec3(self.globalColor);
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]);
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(pkd_closesthit_occlusion)() {
    optixSetPayload_0(1);
}


MM_OPTIX_BOUNDS_KERNEL(pkd_bounds)
(const void* geomData, const float* radData, float radius, box3f& primBounds, const unsigned int primID) {}


MM_OPTIX_INTERSECTION_KERNEL(treelets_intersect)
() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = getProgramData<TreeletsGeoData>();
    const auto treelet = self.treeletBufferPtr[treeletID];

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        {
            auto const ray =
                Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
            if (!clipToBounds(ray, treelet.bounds, t0, t1))
                return;
        }
        auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0),
            fminf(optixGetRayTmax(), t1));

        int nodeID = 0;
        float tmp_hit_t = ray.tmax;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };
        StackEntry stackBase[STACK_DEPTH];
        StackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;
                const PKDParticle& particle = self.particleBufferPtr[particleID];
                int const dim = particle.dim;

                const float t_slab_lo = (particle.pos[dim] - self.radius - org[dim]) * rdir[dim];
                const float t_slab_hi = (particle.pos[dim] + self.radius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(particle, self.radius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;
                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);
                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit)
() {
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<TreeletsGeoData>();

    prd.particleID = primID;
    const PKDParticle& particle = self.particleBufferPtr[primID];
    prd.pos = particle.pos;
    glm::vec3 geo_col = glm::vec3(self.globalColor);
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]);
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(treelets_closesthit_occlusion)
() {
    optixSetPayload_0(1);
}


MM_OPTIX_BOUNDS_KERNEL(treelets_bounds)
(const void* geomData, const float* radData, float radius, box3f& primBounds, const unsigned int primID) {}


//inline __device__ PKDParticle const& decode_coord(QPKDParticle const& coord, glm::vec3 const& center, glm::vec3 const& span) {
//    constexpr unsigned int digits = 1023u;
//    auto const diff = span / static_cast<float>(digits);
//    /*auto pos = glm::vec3(static_cast<float>(coord.x) * span.x, static_cast<float>(coord.y) * span.y,
//        static_cast<float>(coord.z) * span.z);
//    pos = pos + center;*/
//    auto const pos = glm::vec3(fmaf(static_cast<float>(coord.x), diff.x, center.x),
//        fmaf(static_cast<float>(coord.y), diff.y, center.y), fmaf(static_cast<float>(coord.z), diff.z, center.z));
//    PKDParticle p;
//    p.dim = coord.dim;
//    p.pos = pos;
//    return p;
//}

struct QStackEntry {
    float t0, t1;
    unsigned int nodeID;
    box3f bounds;
};

MM_OPTIX_INTERSECTION_KERNEL(comp_treelets_intersect)
() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = getProgramData<QTreeletsGeoData>();
    const auto treelet = self.treeletBufferPtr[treeletID];

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        {
            auto const ray =
                Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
            if (!clipToBounds(ray, treelet.bounds, t0, t1))
                return;
        }
        auto const ray = Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), fmaxf(optixGetRayTmin(), t0),
            fminf(optixGetRayTmax(), t1));

        auto bounds = treelet.bounds;
        glm::vec3 _center;
        glm::vec3 _span;

        int nodeID = 0;
        float tmp_hit_t = ray.tmax;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };
        QStackEntry stackBase[STACK_DEPTH];
        QStackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };

        glm::vec3 tmp_hit_pos;

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down
                _center = bounds.center();
                _span = bounds.span();

                const int particleID = nodeID + begin;
                const PKDParticle& particle = decode_coord(self.particleBufferPtr[particleID], _center, _span);
                int const dim = particle.dim;

                const float t_slab_lo = (particle.pos[dim] - self.radius - org[dim]) * rdir[dim];
                const float t_slab_hi = (particle.pos[dim] + self.radius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(particle, self.radius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                        tmp_hit_pos = particle.pos;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                // we have lB and rB
                // in case of dirSign == 1: near -> rB; far -> lB
                // in case of dirSign == 0: near -> lB; far -> rB

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    stackPtr->bounds = bounds;
                    if (dir_sign[dim]) {
                        // left
                        bounds.upper[dim] = particle.pos[dim] + self.radius;
                    } else {
                        // right
                        bounds.lower[dim] = particle.pos[dim] - self.radius;
                    }
                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;
                    if (dir_sign[dim]) {
                        // right
                        bounds.lower[dim] = particle.pos[dim] - self.radius;
                    } else {
                        // left
                        bounds.upper[dim] = particle.pos[dim] + self.radius;
                    }
                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
                if (need_nearSide) {
                    if (dir_sign[dim]) {
                        // right
                        bounds.lower[dim] = particle.pos[dim] - self.radius;
                    } else {
                        // left
                        bounds.upper[dim] = particle.pos[dim] + self.radius;
                    }
                } else {
                    if (dir_sign[dim]) {
                        // left
                        bounds.upper[dim] = particle.pos[dim] + self.radius;
                    } else {
                        // right
                        bounds.lower[dim] = particle.pos[dim] - self.radius;
                    }
                }
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID, __float_as_uint(tmp_hit_pos.x),
                            __float_as_uint(tmp_hit_pos.y), __float_as_uint(tmp_hit_pos.z));
                        //PerRayData& prd = getPerRayData<PerRayData>();
                        //prd.pos = tmp_hit_pos;
                    }
                    return;
                }
                --stackPtr;
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);
                bounds = stackPtr->bounds;
                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}

MM_OPTIX_CLOSESTHIT_KERNEL(comp_treelets_closesthit)
() {
    const unsigned int primID = optixGetAttribute_0();
    PerRayData& prd = getPerRayData<PerRayData>();

    const auto& self = getProgramData<QTreeletsGeoData>();

    prd.particleID = primID;
    //const PKDParticle& particle = self.particleBufferPtr[primID];
    //prd.pos = particle.pos;
    prd.pos.x = __uint_as_float(optixGetAttribute_1());
    prd.pos.y = __uint_as_float(optixGetAttribute_2());
    prd.pos.z = __uint_as_float(optixGetAttribute_3());
    glm::vec3 geo_col = glm::vec3(self.globalColor);
    if (self.hasColorData) {
        geo_col = glm::vec3(self.colorBufferPtr[primID]);
    }
    prd.albedo = geo_col;
    prd.t = optixGetRayTmax();
    set_depth(prd, optixGetRayTmax());
}


MM_OPTIX_CLOSESTHIT_KERNEL(comp_treelets_closesthit_occlusion)
() {
    optixSetPayload_0(1);
}

} // namespace device
} // namespace optix_hpg
} // namespace megamol
